#include "hip/hip_runtime.h"
#include "kdtree_gpu.hpp"
// clang-format on

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sort.h>

#include "../util/cudaCheckError.hpp"

namespace Sculptor {
namespace {
constexpr int kThreads = 128;
constexpr int kBlocks = 16;
constexpr int kStackDepth = 32;
constexpr float kEps = 0.001f;

struct ScaleFunctor {
  __host__ __device__ int operator()(float x) { return scaling_factor * x; }
  __host__ __device__ float operator()(int x) { return descaling_factor * x; }

 private:
  const int scaling_factor = 2048;
  const float descaling_factor = 1.f / scaling_factor;
};
struct EqualToZero {
  __host__ __device__ bool operator()(const int x) { return x == 0; }
};
struct TupleToVec3 {
  __host__ __device__ glm::vec3 operator()(
      thrust::tuple<float, float, float> const& t) {
    return glm::vec3{t.get<0>(), t.get<1>(), t.get<2>()};
  }
};
struct alignas(int) StackEntry {
  int begin;
  int end;
  struct alignas(char) {
    char level;
    char visited_branch;  // left: -1, none: 0, right: 1
  } misc;
};

void ConstructRecursive(thrust::device_vector<int>& x,
                        thrust::device_vector<int>& y,
                        thrust::device_vector<int>& z,
                        int begin,
                        int end) {
  if (end <= begin)
    return;
  auto mid = begin + (end - begin) / 2;
  auto zip =
      thrust::make_zip_iterator(thrust::make_tuple(y.begin(), z.begin()));
  thrust::sort_by_key(thrust::device, x.begin() + begin, x.begin() + end,
                      zip + begin);
  ConstructRecursive(y, z, x, begin, mid);
  ConstructRecursive(y, z, x, mid + 1, end);
}

__host__ __device__ __forceinline__ float dist2(float3 v,
                                                float x,
                                                float y,
                                                float z) {
  return thrust::max(abs(v.x - x), thrust::max(abs(v.y - y), abs(v.z - z)));
}

__global__ void FindToRemoveKernel(float const* const kd_x,
                                   float const* const kd_y,
                                   float const* const kd_z,
                                   int const kd_size,
                                   float const* const query_points,
                                   int* const should_stay,
                                   float const threshold) {
  __shared__ float s_query_pts[3 * kThreads];
  __shared__ StackEntry stack[kStackDepth];
  __shared__ int stack_top;
  __shared__ int go_left_votes;

  {  // retrieve query point
    int tid = 3 * blockIdx.x * blockDim.x + threadIdx.x;
    s_query_pts[threadIdx.x] = query_points[tid];
    s_query_pts[threadIdx.x + blockDim.x] = query_points[tid + blockDim.x];
    s_query_pts[threadIdx.x + 2 * blockDim.x] =
        query_points[tid + 2 * blockDim.x];
    __syncthreads();
  }
  float3 const query_point =
      reinterpret_cast<float3*>(s_query_pts)[threadIdx.x];
  int cur_nearest_point = kd_size / 2;
  float cur_best_dist = dist2(query_point, kd_x[cur_nearest_point],
                              kd_y[cur_nearest_point], kd_z[cur_nearest_point]);
  if (threadIdx.x == 0)
    stack[stack_top = 0] = {0, kd_size, {0, 0}};

FIND_PROC : {
  __syncthreads();
  if (stack[stack_top].end <= stack[stack_top].begin)
    goto RETURN;
  auto mid = (stack[stack_top].begin + stack[stack_top].end) / 2;
  auto dist_to_mid = dist2(query_point, kd_x[mid], kd_y[mid], kd_z[mid]);
  if (dist_to_mid < cur_best_dist) {
    cur_best_dist = dist_to_mid;
    cur_nearest_point = mid;
  }
  if (stack[stack_top].begin + 1 == stack[stack_top].end)
    goto RETURN;
  auto diff = 0.f;
  switch (stack[stack_top].misc.level) {
    default:
    case 0:
      diff = query_point.x - kd_x[mid];
      break;
    case 1:
      diff = query_point.y - kd_y[mid];
      break;
    case 2:
      diff = query_point.z - kd_z[mid];
      break;
  }
  switch (stack[stack_top].misc.visited_branch) {
    default:
    case 0:
      if (threadIdx.x == 0)
        go_left_votes = 0;
      __syncthreads();
      atomicAdd(&go_left_votes, diff < 0);
      __syncthreads();
      if (threadIdx.x == 0) {
        ++stack_top;
        stack[stack_top] = stack[stack_top - 1];
        if (go_left_votes > blockDim.x / 2) {
          stack[stack_top - 1].misc.visited_branch = -1;
          stack[stack_top].end = mid;
        } else {
          stack[stack_top - 1].misc.visited_branch = 1;
          stack[stack_top].begin = mid + 1;
        }
        stack[stack_top].misc.level = stack[stack_top].misc.level == 2
                                          ? 0
                                          : (stack[stack_top].misc.level + 1);
        stack[stack_top].misc.visited_branch = 0;
      }
      goto FIND_PROC;
    case 1:
      if (__syncthreads_or(diff < kEps)) {
        if (threadIdx.x == 0) {
          stack[stack_top].end = mid;
          stack[stack_top].misc.level = stack[stack_top].misc.level == 2
                                            ? 0
                                            : (stack[stack_top].misc.level + 1);
          stack[stack_top].misc.visited_branch = 0;
        }
        goto FIND_PROC;
      }
      goto RETURN;
    case -1:
      if (__syncthreads_or(diff > -kEps)) {
        if (threadIdx.x == 0) {
          stack[stack_top].begin = mid + 1;
          stack[stack_top].misc.level = stack[stack_top].misc.level == 2
                                            ? 0
                                            : (stack[stack_top].misc.level + 1);
          stack[stack_top].misc.visited_branch = 0;
        }
        goto FIND_PROC;
      }
      goto RETURN;
  }
}
RETURN : {
  if (stack_top > 0) {
    __syncthreads();
    if (threadIdx.x == 0)
      --stack_top;
    goto FIND_PROC;
  }
}
  if (cur_best_dist < threshold)
    should_stay[cur_nearest_point] = 0;
}
}  // namespace

void KdTreeGPU::Construct(float* x, float* y, float* z, int size) {
  thrust::device_vector<int> x_int(size), y_int(size), z_int(size);

  thrust::transform(x, x + size, x_int.begin(), ScaleFunctor());
  thrust::transform(y, y + size, y_int.begin(), ScaleFunctor());
  thrust::transform(z, z + size, z_int.begin(), ScaleFunctor());

  ConstructRecursive(x_int, y_int, z_int, 0, size);

  thrust::transform(x_int.begin(), x_int.end(), x, ScaleFunctor());
  thrust::transform(y_int.begin(), y_int.end(), y, ScaleFunctor());
  thrust::transform(z_int.begin(), z_int.end(), z, ScaleFunctor());
}

std::vector<glm::vec3> KdTreeGPU::RemoveNearest(float* x,
                                                float* y,
                                                float* z,
                                                int kd_size,
                                                float* query_points,
                                                int query_points_size,
                                                float threshold) {
  int* dshould_stay;
  SculptorCudaCheckError(hipMalloc(reinterpret_cast<void**>(&dshould_stay),
                                    sizeof(int) * kd_size));
  thrust::fill(thrust::device_ptr<int>(dshould_stay),
               thrust::device_ptr<int>(dshould_stay) + kd_size, 1);

  int iteration = 0;
  int excess = query_points_size % (kThreads * kBlocks);
  int max = query_points_size - excess;
  std::vector<hipStream_t> streams;
  streams.reserve((max / (kThreads * kBlocks) + 2));
  for (; iteration < max; iteration += kThreads * kBlocks) {
    hipStream_t stream;
    SculptorCudaCheckError(hipStreamCreate(&stream));
    FindToRemoveKernel<<<kBlocks, kThreads, 0, stream>>>(
        x, y, z, kd_size, query_points + 3 * iteration, dshould_stay,
        threshold);
    streams.emplace_back(stream);
  }
  if (excess >= kThreads) {
    hipStream_t stream;
    SculptorCudaCheckError(hipStreamCreate(&stream));
    FindToRemoveKernel<<<excess / kThreads, kThreads, 0, stream>>>(
        x, y, z, kd_size, query_points + 3 * iteration, dshould_stay,
        threshold);
    streams.emplace_back(stream);
  }
  iteration += (excess / kThreads) * kThreads;
  excess %= kThreads;
  if (excess > 0) {
    hipStream_t stream;
    SculptorCudaCheckError(hipStreamCreate(&stream));
    FindToRemoveKernel<<<1, excess, 0, stream>>>(x, y, z, kd_size,
                                                 query_points + 3 * iteration,
                                                 dshould_stay, threshold);
    streams.emplace_back(stream);
  }
  for (auto const& stream : streams) {
    SculptorCudaCheckError(hipStreamSynchronize(stream));
    SculptorCudaCheckError(hipStreamDestroy(stream));
  }
  streams.clear();

  thrust::device_ptr<int> should_stay_dev_ptr(dshould_stay);
  auto xyz = thrust::make_zip_iterator(thrust::make_tuple(
      thrust::device_ptr<float>(x), thrust::device_ptr<float>(y),
      thrust::device_ptr<float>(z)));
  auto to_remove = kd_size - thrust::reduce(should_stay_dev_ptr,
                                            should_stay_dev_ptr + kd_size);
  thrust::device_vector<thrust::tuple<float, float, float>> ret_dev_raw(
      to_remove);
  auto ret_dev_raw_end =
      thrust::copy_if(xyz, xyz + kd_size, should_stay_dev_ptr,
                      ret_dev_raw.begin(), EqualToZero());
  thrust::device_vector<glm::vec3> ret_dev(ret_dev_raw_end -
                                           ret_dev_raw.begin());
  thrust::transform(ret_dev_raw.begin(), ret_dev_raw_end, ret_dev.begin(),
                    TupleToVec3());
  std::vector<glm::vec3> ret(ret_dev_raw_end - ret_dev_raw.begin());
  thrust::copy(ret_dev.begin(), ret_dev.end(), ret.begin());

  thrust::remove_if(xyz, xyz + kd_size, should_stay_dev_ptr, EqualToZero());
  SculptorCudaCheckError(hipFree(dshould_stay));

  return ret;
}
}  // namespace Sculptor
