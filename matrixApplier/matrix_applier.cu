#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include "matrix_applier.hpp"

namespace Sculptor {
namespace {
constexpr int kMatrixSize = 4;
constexpr int kThreads = 128;
constexpr int kBlocks = 32;

__constant__ float c_matrix[kMatrixSize][kMatrixSize];

void __global__ ApplyUnrestrictedKernel(float* vectors, int size) {
  __shared__ float s_data[3 * kThreads];
  int index = 0;
  int off = 3 * kThreads * kBlocks - threadIdx.x - 1;
  size -= off;
  for (index = 3 * kThreads * blockIdx.x + threadIdx.x; index < size;
       index += 3 * kThreads * kBlocks) {
    s_data[threadIdx.x] = vectors[index];
    s_data[threadIdx.x + kThreads] = vectors[index + kThreads];
    s_data[threadIdx.x + 2 * kThreads] = vectors[index + 2 * kThreads];
    __syncthreads();
    float3 v = reinterpret_cast<float3*>(s_data)[threadIdx.x];
    reinterpret_cast<float3*>(s_data)[threadIdx.x] =
        float3{v.x * c_matrix[0][0] + v.y * c_matrix[1][0] +
                   v.z * c_matrix[2][0] + c_matrix[3][0],
               v.x * c_matrix[0][1] + v.y * c_matrix[1][1] +
                   v.z * c_matrix[2][1] + c_matrix[3][1],
               v.x * c_matrix[0][2] + v.y * c_matrix[1][2] +
                   v.z * c_matrix[2][2] + c_matrix[3][2]};
    __syncthreads();
    vectors[index] = s_data[threadIdx.x];
    vectors[index + kThreads] = s_data[threadIdx.x + kThreads];
    vectors[index + 2 * kThreads] = s_data[threadIdx.x + 2 * kThreads];
  }
  size += off;
  __syncthreads();
  // Handle leftover vectors
  if (index < size)
    s_data[threadIdx.x] = vectors[index];
  if (index + kThreads < size)
    s_data[threadIdx.x + kThreads] = vectors[index + kThreads];
  if (index + 2 * kThreads < size)
    s_data[threadIdx.x + 2 * kThreads] = vectors[index + 2 * kThreads];
  __syncthreads();
  float3 v = reinterpret_cast<float3*>(s_data)[threadIdx.x];
  reinterpret_cast<float3*>(s_data)[threadIdx.x] =
      float3{v.x * c_matrix[0][0] + v.y * c_matrix[1][0] +
                 v.z * c_matrix[2][0] + c_matrix[3][0],
             v.x * c_matrix[0][1] + v.y * c_matrix[1][1] +
                 v.z * c_matrix[2][1] + c_matrix[3][1],
             v.x * c_matrix[0][2] + v.y * c_matrix[1][2] +
                 v.z * c_matrix[2][2] + c_matrix[3][2]};
  __syncthreads();
  if (index < size)
    vectors[index] = s_data[threadIdx.x];
  if (index + kThreads < size)
    vectors[index + kThreads] = s_data[threadIdx.x + kThreads];
  if (index + 2 * kThreads < size)
    vectors[index + 2 * kThreads] = s_data[threadIdx.x + 2 * kThreads];
}
}  // namespace
void MatrixApplier::Apply(std::vector<glm::vec3>& vectors,
                          glm::mat4 const& matrix) {
  auto extra_space = vectors.size() % (kThreads * kBlocks);
  if (extra_space != 0)
    extra_space = kThreads * kBlocks - extra_space;

  float* dvectors = nullptr;
  hipMalloc(&dvectors, sizeof(float) * 3 * (vectors.size() + extra_space));
  hipMemcpy(dvectors, vectors.data(), sizeof(float) * 3 * vectors.size(),
             hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(c_matrix), &matrix, sizeof(c_matrix), 0);
  ApplyUnrestrictedKernel<<<kBlocks, kThreads>>>(
      dvectors, 3 * (vectors.size() + extra_space));
  hipMemcpy(vectors.data(), dvectors, sizeof(float) * 3 * vectors.size(),
             hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  hipFree(dvectors);
}
void MatrixApplier::Apply(hipGraphicsResource* vectors,
                          int nvectors,
                          glm::mat4 const& matrix) {
  hipGraphicsMapResources(1, &vectors);
  float* dvectors = nullptr;
  size_t num_bytes;
  hipMemcpyToSymbol(HIP_SYMBOL(c_matrix), &matrix, sizeof(c_matrix), 0);
  hipGraphicsResourceGetMappedPointer(reinterpret_cast<void**>(&dvectors),
                                       &num_bytes, vectors);

  ApplyUnrestrictedKernel<<<kBlocks, kThreads>>>(dvectors, 3 * nvectors);
  hipDeviceSynchronize();

  hipGraphicsUnmapResources(1, &vectors);
}
}  // namespace Sculptor
